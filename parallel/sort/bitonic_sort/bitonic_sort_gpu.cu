#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS 512 // 2^9
#define BLOCKS 32768 // 2^15
#define NUM_VALS THREADS*BLOCKS

__device__ void swap(unsigned int a, unsigned int b, float *data){
	float temp = data[a];
	data[a]=data[b];
	data[b]=temp;
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  extern __shared__ float buf[];

  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i]>dev_values[ixj]) {
        /* exchange(i,ixj); */
        swap(i, ixj, dev_values);
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i]<dev_values[ixj]) {
        /* exchange(i,ixj); */
        swap(i, ixj, dev_values);
      }
    }
  }
}

void bitonic_sort(float *values, int count)
{
  float *dev_values;
  size_t size = count * sizeof(float);

  hipMalloc((void**) &dev_values, size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */

  int j, k;
  /* Major step */
  for (k = 2; k <= count; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
    }
  }
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  hipFree(dev_values);
}

__global__ void D_bitonic_sort_K(float *data_to_sort)
{
    extern __shared__ float buf[];
    uint desc; // bit position that determines sort order ascending/descending
	uint stride; // distance to partner index
	int thread = threadIdx.x;

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    buf[thread] = data_to_sort[thread];
    __syncthreads();

	for (desc=2; desc<=blockDim.x; desc*=2) {
		for (stride = desc>>1; stride>0; stride=stride>>1) {
			int partner = idx ^ stride;
			if (partner > idx) {
				if ( (thread & desc) == 0 && data_to_sort[idx] > data_to_sort[partner] ) {
					swap(idx,partner,data_to_sort);
				} else if ( (thread & desc) != 0 && data_to_sort[idx]< data_to_sort[partner] ) {
					swap(idx,partner,data_to_sort);
				}
            }
            __syncthreads();
		}
	}
}

int main() {
    float* data = (float*)malloc(NUM_VALS * sizeof(float));
    bitonic_sort(data, NUM_VALS);
}